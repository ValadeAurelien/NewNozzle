#include "hip/hip_runtime.h"
typedef float data_t;

struct error
{
  char mess[64];

  __host__ __device__
  error() {}
  __host__ __device__
  error(char* _mess) 
  { 
    int i=0;
    do {
      mess[i] = _mess[i];
    } while (mess[i++] !=0 && i<63);
  }

  __host__ __device__
  char*  what() { return mess; }
};

//struct vec2D_t
//{
//  data_t a, b;
//
//  __host__ __device__
//  vec2D_t(data_t _a, data_t _b) : a(_a), b(_b) {}
////  __host__ __device__
////  vec2D_t(const vec2D_t& v) : a(v.a), b(v.b) {}
//
//  __host__ __device__
//  vec2D_t operator+(const vec2D_t& v) { return vec2D_t(a+v.a, b+v.b); }
//  __host__ __device__
//  vec2D_t operator-(const vec2D_t& v) { return vec2D_t(a-v.a, b-v.b); }
//  __host__ __device__
//  vec2D_t operator*(const data_t& f) { return vec2D_t(a*f, b*f); }
//  __host__ __device__
//  vec2D_t operator/(const data_t& f) { return vec2D_t(a/f, b/f); }
// 
//  __host__ __device__
//  data_t scalar(const vec2D_t& v) { return a*v.a+b*v.b; }
//  __host__ __device__
//  data_t norm() { return sqrt(a*a+b*b); }
//};

struct cell_t
{
  data_t vr, vz, P, rho, T;
  bool is_wall;

  __host__ __device__
  cell_t(data_t _vr, data_t _vz, data_t _P, data_t _rho, data_t _T, bool _is_wall) : vr(_vr), vz(_vz), P(_P), rho(_rho), T(_T), is_wall(_is_wall) {}
//  __host__ __device__
//  cell_t(const cell_t& c) : v(c.v), P(c.P), rho(c.rho), T(c.T) {}

  __host__ __device__
  cell_t operator+(const cell_t& c) { return cell_t(vr+c.vr, vz+c.vz, P+c.P, rho+c.rho, T+c.T, is_wall); }
  __host__ __device__
  cell_t operator-(const cell_t& c) { return cell_t(vr-c.vr, vz-c.vz, P-c.P, rho-c.rho, T-c.T, is_wall); }
  __host__ __device__
  cell_t operator*(const data_t& f) { return cell_t(vr*f, vz*f, P*f, rho*f, T*f, is_wall); }
  __host__ __device__
  cell_t operator/(const data_t& f) { return cell_t(vr/f, vz/f, P/f, rho/f, T/f, is_wall); }
};

__global__
void cuda_operator_plus(cell_t* C, cell_t* A, cell_t* B, int size_i, int size_j)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i>=size_i || j>=size_j) return;
  C[j*size_i+i] = A[j*size_i+i] + B[j*size_i+i];
}
__global__
void cuda_operator_minus(cell_t* C, cell_t* A, cell_t* B, int size_i, int size_j)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i>=size_i || j>=size_j) return;
  C[j*size_i+i] = A[j*size_i+i] - B[j*size_i+i];
}
__global__
void cuda_operator_times(cell_t* B, cell_t* A, data_t f, int size_i, int size_j)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i>=size_i || j>=size_j) return;
  B[j*size_i+i] = A[j*size_i+i] * f;
}
__global__
void cuda_operator_divided(cell_t* B, cell_t* A, data_t f, int size_i, int size_j)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i>=size_i || j>=size_j) return;
  B[j*size_i+i] = A[j*size_i+i] / f;
}

struct meshgrid_t
{
  int size_i, size_j;
  dim3 gridsize, blocksize;
  cell_t* h_data, *d_data;

  __host__  
  meshgrid_t() {}
  __host__  
  meshgrid_t(int _size_i, int _size_j, dim3 _gridsize, dim3 _blocksize) :
    size_i(_size_i), size_j(_size_j), gridsize(_gridsize), blocksize(_blocksize)
  {
    hipMalloc(&d_data, sizeof(cell_t)*size_i*size_j);
  }
  __host__ 
  ~meshgrid_t()
  {
    hipFree(&d_data);
    if (h_data) free(h_data);
  }
  __host__  
  from_meshgrid_prop(const meshgrid_t& m) 
  {
    size_i = m.size_i;
    size_j = m.size_j;
    gridsize = m.gridsize;
    blocksize = m.blocksize;
    if (!d_data) hipMalloc(&d_data, sizeof(cell_t)*size_i*size_j);
  }

  __host__
  meshgrid_t operator+(const meshgrid_t& m) const
  { 
    if (!equivalent(m)) throw error("meshgrids not equiarlent"); 
    meshgrid_t res(size_i, size_j, gridsize, blocksize);
    cuda_operator_plus<<<blocksize, gridsize>>>(res.d_data, d_data, m.d_data, size_i, size_j);
    return res;
  }
  __host__
  meshgrid_t operator-(const meshgrid_t& m) const 
  { 
    if (!equivalent(m)) throw error("meshgrids not equivalent"); 
    meshgrid_t res(size_i, size_j, gridsize, blocksize);
    cuda_operator_minus<<<blocksize, gridsize>>>(res.d_data, d_data, m.d_data, size_i, size_j);
    return res;
  }

  __host__
  meshgrid_t operator*(const data_t& f) const 
  { 
    meshgrid_t res(size_i, size_j, gridsize, blocksize);
    cuda_operator_times<<<blocksize, gridsize>>>(res.d_data, d_data, f, size_i, size_j);
    return res;
  }
  __host__
  meshgrid_t operator/(const data_t& f) const 
  { 
    meshgrid_t res(size_i, size_j, gridsize, blocksize);
    cuda_operator_divided<<<blocksize, gridsize>>>(res.d_data, d_data, f, size_i, size_j);
    return res;
  }

  __host__ 
  data_t compare(const meshgrid_t& m) const
  {
    return 0;
//    cell_t *max_diff, *max_m;
//    hipMalloc(&max_diff, sizeof(cell_t));
//    cudamaxdiff<<<blocksize, gridsize>>>(max_diff, d_data, m.d_data, size_i, size_j);
  }
  __host__ 
  bool equivalent(const meshgrid_t& m) const
  {
    return (size_i==m.size_i && size_j==m.size_j && 
            gridsize.x==m.gridsize.x && blocksize.x==m.blocksize.x &&
            gridsize.y==m.gridsize.y && blocksize.y==m.blocksize.y);
  }

  __host__ 
  void to_host()
  {
    h_data = (cell_t*) malloc(sizeof(cell_t)*size_i*size_j);
    hipMemcpy(h_data, d_data, sizeof(cell_t)*size_i*size_j, hipMemcpyDeviceToHost); 
  }
  __host__ 
  const cell_t& at(int i, int j) const
  {
    if (i<size_i && j<size_j)
      return h_data[j*size_i+i];
    else 
      throw error("out of range");
  }
  __device__ 
  const cell_t& dat(int i, int j) const
  {
    if (i<size_i && j<size_j)
      return d_data[j*size_i+i];
    else 
      return d_data[0];
  }
};


